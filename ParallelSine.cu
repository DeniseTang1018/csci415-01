#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s):Chengyao Tang,Victoria Kyereme  
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information
__global__ void sine_parallel(float*d_input,float*d_output ){
	int idx = threadIdx.x;
	float d_value = d_input[idx];
	float d_numer = d_input[idx]*d_input[idx]*d_input[idx];
	int   d_denom = 6;
	int   d_sign = -1;
	for (int d_j=1;d_j<=TERMS; d_j++){
		d_value += d_sign *d_numer/d_denom;
		d_numer *= d_input[idx]* d_input[idx];
		d_denom *= (2*d_j+2)* (2*d_j+3);
		d_sign *= -1; 
}
		d_output[idx] = d_value;
}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  float *h_gpu_result = (float*)malloc(N*sizeof(float));
 
 //declare GPU memory pointers
  float *d_input;
  float *d_output;
  
  long long Memory_Allocation_start_time = start_timer();
  long long GPU_start_time = start_timer();
  //allocate GPU memory
  hipMalloc((void **) &d_input,N*sizeof(float));
  hipMalloc((void **) &d_output,N*sizeof(float));
  long long Memory_Allocation_end_time = stop_timer(Memory_Allocation_start_time,"\nGPU Memory allocation time:");

  //transfer the array to the GPU
  long long Memory_copy_to_device_start_time = start_timer();
  hipMemcpy(d_input, h_input, N*sizeof(float),hipMemcpyHostToDevice);;
  long long Memory_copy_to_device_end_time = stop_timer(Memory_copy_to_device_start_time,"\nGPU Memory Copy to Device time:");
 
  //launch the kernel
  int threards = N/1024;
  long long Kernal_run_start_time = start_timer();
  sine_parallel<<<threards,1024>>>(d_input,d_output);
  long long Kernal_run_end_time = stop_timer(Kernal_run_start_time,"\nGPU Kernal run Time:");

  //copy back the result array to the CPU
  long long Memory_copy_to_Host_start_time = start_timer();
  hipMemcpy(h_gpu_result,d_output,N*sizeof(float),hipMemcpyDeviceToHost);
  long long Memory_copy_to_Host_end_time = stop_timer(Memory_copy_to_Host_start_time,"\nGPU Memory Copy to Host time:");
  long long GPU_end_time = stop_timer(GPU_start_time,"\nTotal GPU Run time:");
 

  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






